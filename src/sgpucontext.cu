#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 *
 * Code and text by Sean Baxter, NVIDIA Research
 * See http://nvlabs.github.io/moderngpu for repository and documentation.
 *
 ******************************************************************************/

#include "util/sgpucontext.h"
#include "util/format.h"

namespace sgpu {

////////////////////////////////////////////////////////////////////////////////
// CudaTimer

void CudaTimer::Start() {
	hipEventRecord(start);
	hipDeviceSynchronize();
}
double CudaTimer::Split() {
	hipEventRecord(end);
	hipDeviceSynchronize();
	float t;
	hipEventElapsedTime(&t, start, end);
	start.Swap(end);
	return (t / 1000.0);
}
double CudaTimer::Throughput(int count, int numIterations) {
	double elapsed = Split();
	return (double)numIterations * count / elapsed;
}

////////////////////////////////////////////////////////////////////////////////
// CudaDevice

__global__ void KernelVersionShim() { }

struct DeviceGroup {
	int numCudaDevices;
	CudaDevice** cudaDevices;

	DeviceGroup() {
		numCudaDevices = -1;
		cudaDevices = 0;
	}

	int GetDeviceCount() {
		if(-1 == numCudaDevices) {
			hipError_t error = hipGetDeviceCount(&numCudaDevices);
			if(hipSuccess != error || numCudaDevices <= 0) {
				fprintf(stderr, "ERROR ENUMERATING CUDA DEVICES.\nExiting.\n");
				exit(0);
			}
			cudaDevices = new CudaDevice*[numCudaDevices];
			memset(cudaDevices, 0, sizeof(CudaDevice*) * numCudaDevices);
		}
		return numCudaDevices;
	}

	CudaDevice* GetByOrdinal(int ordinal) {
		if(ordinal >= GetDeviceCount()) return 0;

		if(!cudaDevices[ordinal]) {
			// Retrieve the device properties.
			CudaDevice* device = cudaDevices[ordinal] = new CudaDevice;
			device->_ordinal = ordinal;
			hipError_t error = hipGetDeviceProperties(&device->_prop,
				ordinal);
			if(hipSuccess != error) {
				fprintf(stderr, "FAILURE TO CREATE CUDA DEVICE %d\n", ordinal);
				exit(0);
			}

			// Get the compiler version for this device.
			hipSetDevice(ordinal);
			hipFuncAttributes attr;
			error = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(KernelVersionShim));
			if(hipSuccess == error)
				device->_ptxVersion = 10 * attr.ptxVersion;
			else {
				printf("NOT COMPILED WITH COMPATIBLE PTX VERSION FOR DEVICE"
					" %d\n", ordinal);
				// The module wasn't compiled with support for this device.
				device->_ptxVersion = 0;
			}
		}
		return cudaDevices[ordinal];
	}

	~DeviceGroup() {
		if(cudaDevices) {
			for(int i = 0; i < numCudaDevices; ++i)
				delete cudaDevices[i];
			delete [] cudaDevices;
		}
		hipDeviceReset();
	}
};

std::auto_ptr<DeviceGroup> deviceGroup;


int CudaDevice::DeviceCount() {
	if(!deviceGroup.get())
		deviceGroup.reset(new DeviceGroup);
	return deviceGroup->GetDeviceCount();
}

CudaDevice& CudaDevice::ByOrdinal(int ordinal) {
	if(ordinal < 0 || ordinal >= DeviceCount()) {
		fprintf(stderr, "CODE REQUESTED INVALID CUDA DEVICE %d\n", ordinal);
		exit(0);
	}
	return *deviceGroup->GetByOrdinal(ordinal);
}

CudaDevice& CudaDevice::Selected() {
	int ordinal;
	hipError_t error = hipGetDevice(&ordinal);
	if(hipSuccess != error) {
		fprintf(stderr, "ERROR RETRIEVING CUDA DEVICE ORDINAL\n");
		exit(0);
	}
	return ByOrdinal(ordinal);
}

void CudaDevice::SetActive() {
	hipError_t error = hipSetDevice(_ordinal);
	if(hipSuccess != error) {
		fprintf(stderr, "ERROR SETTING CUDA DEVICE TO ORDINAL %d\n", _ordinal);
		exit(0);
	}
}

std::string CudaDevice::DeviceString() const {
	size_t freeMem, totalMem;
	hipError_t error = hipMemGetInfo(&freeMem, &totalMem);
	if(hipSuccess != error) {
		fprintf(stderr, "ERROR RETRIEVING MEM INFO FOR CUDA DEVICE %d\n",
			_ordinal);
		exit(0);
	}

	double memBandwidth = (_prop.memoryClockRate * 1000.0) *
		(_prop.memoryBusWidth / 8 * 2) / 1.0e9;

	std::string s = stringprintf(
		"%s : %8.3lf Mhz   (Ordinal %d)\n"
		"%d SMs enabled. Compute Capability sm_%d%d\n"
		"FreeMem: %6dMB   TotalMem: %6dMB   %2d-bit pointers.\n"
		"Mem Clock: %8.3lf Mhz x %d bits   (%5.1lf GB/s)\n"
		"ECC %s\n\n",
		_prop.name, _prop.clockRate / 1000.0, _ordinal,
		_prop.multiProcessorCount, _prop.major, _prop.minor,
		(int)(freeMem / (1<< 20)), (int)(totalMem / (1<< 20)), 8 * sizeof(int*),
		_prop.memoryClockRate / 1000.0, _prop.memoryBusWidth, memBandwidth,
		_prop.ECCEnabled ? "Enabled" : "Disabled");
	return s;
}

////////////////////////////////////////////////////////////////////////////////
// CudaContext

struct ContextCache {
	CudaContext** standardContexts;
	int numDevices;

	ContextCache() {
		numDevices = CudaDevice::DeviceCount();
		standardContexts = new CudaContext*[numDevices];
		memset(standardContexts, 0, sizeof(CudaContext*) * numDevices);
	}

	CudaContext* GetByOrdinal(int ordinal) {
		if(!standardContexts[ordinal]) {
			CudaDevice& device = CudaDevice::ByOrdinal(ordinal);
			standardContexts[ordinal] = new CudaContext(device, false);
		}
		return standardContexts[ordinal];
	}

	~ContextCache() {
		if(standardContexts) {
			for(int i = 0; i < numDevices; ++i)
				delete standardContexts[i];
			delete [] standardContexts;
		}
	}
};
std::auto_ptr<ContextCache> contextCache;

CudaContext::CudaContext(CudaDevice& device, bool newStream) :
	_event(hipEventDisableTiming /*| hipEventBlockingSync */),
	_stream(0), _noRefCount(false), _pageLocked(0) {

	// Create an allocator.
	_alloc.reset(new CudaAllocSimple(device));

	if(newStream) hipStreamCreate(&_stream);
	_ownStream = newStream;

	// Allocate 4KB of page-locked memory.
	hipError_t error = hipHostMalloc((void**)&_pageLocked, 4096);

	// Allocate an auxiliary stream.
	error = hipStreamCreate(&_auxStream);
}

CudaContext::~CudaContext() {
	if(_pageLocked)
		hipHostFree(_pageLocked);
	if(_ownStream && _stream)
		hipStreamDestroy(_stream);
	if(_auxStream)
		hipStreamDestroy(_auxStream);
}

CudaContext& CudaContext::CachedContext(int ordinal) {
	bool setActive = -1 != ordinal;
	if(-1 == ordinal) {
		hipError_t error = hipGetDevice(&ordinal);
		if(hipSuccess != error) {
			fprintf(stderr, "ERROR RETRIEVING CUDA DEVICE ORDINAL\n");
			exit(0);
		}
	}
	int numDevices = CudaDevice::DeviceCount();

	if(ordinal < 0 || ordinal >= numDevices) {
		fprintf(stderr, "CODE REQUESTED INVALID CUDA DEVICE %d\n", ordinal);
		exit(0);
	}

	CudaContext& context = *contextCache->GetByOrdinal(ordinal);
	if(!context.PTXVersion()) {
		fprintf(stderr, "This CUDA executable was not compiled with support"
			" for device %d (sm_%2d)\n", ordinal, context.ArchVersion() / 10);
		exit(0);
	}

	if(setActive) context.SetActive();
	return context;
}

ContextPtr CreateCudaDevice(int ordinal) {
	CudaDevice& device = CudaDevice::ByOrdinal(ordinal);
	ContextPtr context(new CudaContext(device, false));
	return context;
}

ContextPtr CreateCudaDeviceStream(int ordinal) {
	ContextPtr context(new CudaContext(CudaDevice::ByOrdinal(ordinal), true));
	return context;
}

ContextPtr CreateCudaDeviceAttachStream(int ordinal, hipStream_t stream) {
	ContextPtr context(new CudaContext(CudaDevice::ByOrdinal(ordinal), false));
	context->_stream = stream;
	return context;
}

ContextPtr CreateCudaDeviceAttachStream(hipStream_t stream) {
	int ordinal;
	hipGetDevice(&ordinal);
	return CreateCudaDeviceAttachStream(ordinal, stream);
}

////////////////////////////////////////////////////////////////////////////////
// CudaAllocSimple

hipError_t CudaAllocSimple::Malloc(size_t size, void** p) {
	hipError_t error = hipSuccess;
	*p = 0;
	if(size) error = hipMalloc(p, size);

	if(hipSuccess != error) {
		printf("CUDA MALLOC ERROR %d\n", error);
		exit(0);
	}

	return error;
}

bool CudaAllocSimple::Free(void* p) {
	hipError_t error = hipSuccess;
	if(p) error = hipFree(p);
	return hipSuccess == error;
}

} // namespace sgpu
